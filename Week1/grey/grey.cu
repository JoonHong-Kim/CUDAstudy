#include "hip/hip_runtime.h"

/*
Brutally copied from https ://github.com/junstar92/parallel_programming_study/blob/master/CUDA/imageProcessing/convertColorToGrey.cu
Thank you Junstar92!

sudo apt-get install libopencv-dev

Also have a look at https://github.com/mhezarei/CUDA-RGB-grey

*Compile : nvcc -o grey grey.cu -I.. -lcuda $(pkg-config opencv4 --libs --cflags)
* Run : ./grey < image file path>
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#include <opencv2/core/core.hpp>
#include <opencv2/opencv.hpp>

#define CHANNELS 3

void Usage(char prog_name[])
{
    fprintf(stderr, "Usage: %s <image file path>\n", prog_name);
    exit(EXIT_FAILURE);
}

// Input image has 3 channels corresponding to RGB
// The input image is encoded as unsigned characters [0, 255]
__global__ void colorToGreyscaleConversion(unsigned char *in, unsigned char *out, int width, int height)
{
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    if (Row < height && Col < width)
    {
        int offset = Row * width + Col;
        int rgbOffset = offset * CHANNELS;

        unsigned char r = in[rgbOffset];     // red value for pixel
        unsigned char g = in[rgbOffset + 1]; // green value for pixel
        unsigned char b = in[rgbOffset + 2]; // blue value for pixel

        out[offset] = 0.21f * r + 0.71f * g + 0.07f * b;
    }
}

int main(int argc, char **argv)
{
    if (argc != 2)
    {
        Usage(argv[0]);
    }

    const char *file_name = argv[1];
    int width, height, channels;
    unsigned char *h_origImg, *h_resultImg;
    // open image file
    cv::Mat origImg = cv::imread(file_name);

    width = origImg.cols;
    height = origImg.rows;
    channels = origImg.channels();
    printf("Image size = (%d x %d x %d)\n", width, height, channels);
    assert(channels == CHANNELS);

    cv::Mat half;
    cv::resize(origImg, half, cv::Size(width / 2, height / 2));

    h_origImg = (unsigned char *)malloc(width * height * channels * sizeof(unsigned char));
    h_resultImg = (unsigned char *)malloc(width * height * sizeof(unsigned char));
    (void)memcpy(h_origImg, origImg.data, width * height * channels);

    unsigned char *d_origImg, *d_resultImg;
    hipMalloc((void **)&d_origImg, width * height * channels * sizeof(unsigned char));
    hipMalloc((void **)&d_resultImg, width * height * sizeof(unsigned char));

    // Copy the host input in host memory to the device input in device memory
    hipMemcpy(d_origImg, h_origImg, width * height * channels * sizeof(unsigned char), hipMemcpyHostToDevice);

    // Launch the Kernel
    const int block_size = 16;
    dim3 threads(block_size, block_size);
    dim3 grid(ceil(width / (double)threads.x), ceil(height / (double)threads.y));
    colorToGreyscaleConversion<<<grid, threads>>>(d_origImg, d_resultImg, width, height);

    // Copy the device result in device memory to the host result in host memory
    hipMemcpy(h_resultImg, d_resultImg, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    cv::Mat resultImg(height, width, CV_8UC1);
    memcpy(resultImg.data, h_resultImg, width * height);

    // Free device global memory
    hipFree(d_origImg);
    hipFree(d_resultImg);

    // Free host memory
    free(h_origImg);
    free(h_resultImg);

    // cv::Mat resizeImg;
    cv::resize(resultImg, resultImg, cv::Size(width / 2, height / 2));
    // save image to ./grey.jpg
    cv::imwrite("grey.jpg", resultImg);

    return 0;
}
