#include <bits/stdc++.h>
#include <hip/hip_runtime.h>

#define N_FEATURE 1024
#define N_FIELD 16
#define DIM_FEAT 4
#define INPUT_CNT 1024
#define TILE_SIZE 16

__global__ void betterFM(float *result, float *feature, int *input_data, float *interaction_weight)
{
    __shared__ int s_A[INPUT_CNT];
    int data_idx = blockIdx.x * blockDim.x + threadIdx.x;
    s_A[data_idx] = input_data[data_idx * N_FIELD + 15];
    // __syncthreads();
    float res = 0;
    int didx=0;
    int djdx =0;
    for (int i = 0; i < TILE_SIZE; i++)
    {
        for (int j = i + 1; j < TILE_SIZE; j++)
        {
            didx = input_data[data_idx * TILE_SIZE + i];
            if (j == TILE_SIZE - 1)
            {
                djdx = s_A[data_idx];
            }
            else{
                djdx = input_data[data_idx * TILE_SIZE + j];
            }
            float sum = 0;
            for (int k = 0; k < DIM_FEAT; k++)
            {
                sum += feature[didx * DIM_FEAT + k] * feature[djdx * DIM_FEAT + k];
            }
            res += interaction_weight[i * N_FIELD + j] * sum;
        }
    }
    result[data_idx] = res;
}
float rand_float()
{
    return (float)rand() / (float)RAND_MAX;
}

int main()
{

    // set random seed
    srand(1);

    float *feature;
    float *interaction_weight;

    int *input_data;
    float *result;

    // set feature

    hipMallocManaged(&feature, N_FEATURE * DIM_FEAT * sizeof(float));
    hipMallocManaged(&interaction_weight, N_FIELD * N_FIELD * sizeof(float));

    // randomly set feature
    for (int i = 0; i < N_FEATURE; i++)
    {
        for (int j = 0; j < DIM_FEAT; j++)
        {
            feature[i * DIM_FEAT + j] = rand_float();
        }
    }

    // randomly set interaction weight
    for (int i = 0; i < N_FIELD; i++)
    {
        for (int j = i + 1; j < N_FIELD; j++)
        {
            interaction_weight[i * N_FIELD + j] = rand_float();
        }
    }

    // set input data

    hipMallocManaged(&input_data, INPUT_CNT * N_FIELD * sizeof(int));

    // randomly set input data.
    for (int i = 0; i < INPUT_CNT * N_FIELD; i++)
    {
        input_data[i] = rand() % N_FEATURE;
    }

    hipMallocManaged(&result, INPUT_CNT * sizeof(float));
    dim3 block(16);
    dim3 grid(ceil(N_FEATURE / block.x));
    // run, measure time
    float start = clock();
    for (int i = 0; i < 10000; i++)
    {
        betterFM<<<grid, block>>>(result, feature, input_data, interaction_weight);
        hipDeviceSynchronize();
    }
    float end = clock();
    float time = (end - start) / CLOCKS_PER_SEC;
    // dim3 block(16, 1024);
    // dim3 grid(ceil(INPUT_CNT / block.x));
    // // run, measure time
    // float start = clock();
    // for (int i = 0; i < 1; i++)
    // {
    //     betterFM<<<grid, block>>>(result, feature, input_data, interaction_weight);
    //     cudaDeviceSynchronize();
    // }
    // float end = clock();
    // float time = (end - start) / CLOCKS_PER_SEC;
    printf("time: %f\n", time);

    // check result
    printf("Output:\n");
    for (int i = 0; i < 10; i++)
    {
        printf("result %d, %f\n", i, result[i]);
    }
    printf("result %d, %f\n", INPUT_CNT - 1, result[INPUT_CNT - 1]);

    hipFree(feature);
    hipFree(interaction_weight);
    hipFree(input_data);
    hipFree(result);
    return 0;
}
